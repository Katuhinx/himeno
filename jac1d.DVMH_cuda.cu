#include "hip/hip_runtime.h"
#include <cassert>

#include <cdvmh_helpers.h>



__global__ void loop_jac1d_29_cuda_kernel_int(double * DVMH_RESTRICT A, double * DVMH_RESTRICT B, int boundsLow_1, int boundsHigh_1, int blockOffset) {
    /* Parameters */
    /* Supplementary variables for loop handling */
    int restBlocks, curBlocks;
    /* User variables - loop index variables and other private variables */
    int i;

    restBlocks = blockIdx.x + blockOffset;
    curBlocks = restBlocks;
    i = boundsLow_1 + (curBlocks * blockDim.x + threadIdx.x);
    if (i <= boundsHigh_1)
    {
        do
        {
            A[i] = 0;
            if (i == 0 || i == 8 - 1)
                B[i] = 0;
            else
                B[i] = 2 + i;
        }
        while(0);
    }
}
__global__ void loop_jac1d_29_cuda_kernel_llong(double * DVMH_RESTRICT A, double * DVMH_RESTRICT B, long long boundsLow_1, long long boundsHigh_1, long long blockOffset) {
    /* Parameters */
    /* Supplementary variables for loop handling */
    long long restBlocks, curBlocks;
    /* User variables - loop index variables and other private variables */
    int i;

    restBlocks = blockIdx.x + blockOffset;
    curBlocks = restBlocks;
    i = boundsLow_1 + (curBlocks * blockDim.x + threadIdx.x);
    if (i <= boundsHigh_1)
    {
        do
        {
            A[i] = 0;
            if (i == 0 || i == 8 - 1)
                B[i] = 0;
            else
                B[i] = 2 + i;
        }
        while(0);
    }
}

extern "C" void loop_jac1d_29_cuda(DvmType *pLoopRef, DvmType A_hdr[], DvmType B_hdr[]) {
    DvmType tmpVar;
    /* Loop reference and device number */
    DvmType loop_ref = *pLoopRef;
    DvmType device_num = dvmh_loop_get_device_num_C(loop_ref);
    /* Parameters */
    dvmh_loop_autotransform_C(loop_ref, A_hdr);
    double *A = (double *)dvmh_get_natural_base_C(device_num, A_hdr);
    DvmType A_devHdr[64];
    tmpVar = dvmh_fill_header_C(device_num, A, A_hdr, A_devHdr, 0);
    assert(tmpVar == 0 || tmpVar == 1);
    dvmh_loop_autotransform_C(loop_ref, B_hdr);
    double *B = (double *)dvmh_get_natural_base_C(device_num, B_hdr);
    DvmType B_devHdr[64];
    tmpVar = dvmh_fill_header_C(device_num, B, B_hdr, B_devHdr, 0);
    assert(tmpVar == 0 || tmpVar == 1);
    /* Supplementary variables for loop handling */
    DvmType boundsLow[1], boundsHigh[1], loopSteps[1];
    DvmType blocksS[1];
    DvmType restBlocks;
    dim3 blocks(1, 1, 1), threads(0, 0, 0);
    hipStream_t stream;

    /* Choose index type for CUDA kernel */
    int kernelIndexT = dvmh_loop_guess_index_type_C(loop_ref);
    if (kernelIndexT == rt_LONG) kernelIndexT = (sizeof(long) <= sizeof(int) ? rt_INT : rt_LLONG);
    assert(kernelIndexT == rt_INT || kernelIndexT == rt_LLONG);

    /* Get CUDA configuration parameters */
    extern DvmType loop_jac1d_29_cuda_kernel_int_regs;
    if (kernelIndexT == rt_INT) dvmh_loop_cuda_get_config_C(loop_ref, 0, loop_jac1d_29_cuda_kernel_int_regs, &threads, &stream, 0);
    extern DvmType loop_jac1d_29_cuda_kernel_llong_regs;
    if (kernelIndexT == rt_LLONG) dvmh_loop_cuda_get_config_C(loop_ref, 0, loop_jac1d_29_cuda_kernel_llong_regs, &threads, &stream, 0);

    /* Calculate computation distribution parameters */
    dvmh_loop_fill_bounds_C(loop_ref, boundsLow, boundsHigh, loopSteps);
    blocksS[0] = ((boundsHigh[0] - boundsLow[0] + loopSteps[0]) / loopSteps[0] + (threads.x - 1)) / threads.x;

    /* GPU execution */
    restBlocks = blocksS[0];
    while (restBlocks > 0) {
        blocks.x = (restBlocks <= 65535 ? restBlocks : (restBlocks <= 65535 * 2 ? restBlocks / 2 : 65535));
        if (kernelIndexT == rt_INT) loop_jac1d_29_cuda_kernel_int<<<blocks, threads, 0, stream>>>(A, B, boundsLow[0], boundsHigh[0], blocksS[0] - restBlocks);
        if (kernelIndexT == rt_LLONG) loop_jac1d_29_cuda_kernel_llong<<<blocks, threads, 0, stream>>>(A, B, boundsLow[0], boundsHigh[0], blocksS[0] - restBlocks);
        restBlocks -= blocks.x;
    }
}

__global__ void loop_jac1d_50_cuda_kernel_int(double * DVMH_RESTRICT A, double * DVMH_RESTRICT B, int boundsLow_1, int boundsHigh_1, double eps, double eps_grid[], int blockOffset) {
    /* Parameters */
    /* Supplementary variables for loop handling */
    int restBlocks, curBlocks;
    /* User variables - loop index variables and other private variables */
    int i;

    restBlocks = blockIdx.x + blockOffset;
    curBlocks = restBlocks;
    i = boundsLow_1 + (curBlocks * blockDim.x + threadIdx.x);
    if (i <= boundsHigh_1)
    {
        do
        {
            double tmp = fabs(B[i] - A[i]);
            eps = ((tmp) > (eps) ? (tmp) : (eps));
            A[i] = B[i];
        }
        while(0);
    }

    /* Write reduction values to global memory */
    eps_grid[threadIdx.x + blockDim.x * (threadIdx.y + blockDim.y * (threadIdx.z + blockDim.z * (blockIdx.x + blockOffset)))] = eps;
}
__global__ void loop_jac1d_50_cuda_kernel_llong(double * DVMH_RESTRICT A, double * DVMH_RESTRICT B, long long boundsLow_1, long long boundsHigh_1, double eps, double eps_grid[], long long blockOffset) {
    /* Parameters */
    /* Supplementary variables for loop handling */
    long long restBlocks, curBlocks;
    /* User variables - loop index variables and other private variables */
    int i;

    restBlocks = blockIdx.x + blockOffset;
    curBlocks = restBlocks;
    i = boundsLow_1 + (curBlocks * blockDim.x + threadIdx.x);
    if (i <= boundsHigh_1)
    {
        do
        {
            double tmp = fabs(B[i] - A[i]);
            eps = ((tmp) > (eps) ? (tmp) : (eps));
            A[i] = B[i];
        }
        while(0);
    }

    /* Write reduction values to global memory */
    eps_grid[threadIdx.x + blockDim.x * (threadIdx.y + blockDim.y * (threadIdx.z + blockDim.z * (blockIdx.x + blockOffset)))] = eps;
}

extern "C" void loop_jac1d_50_cuda(DvmType *pLoopRef, DvmType A_hdr[], DvmType B_hdr[]) {
    DvmType tmpVar;
    /* Loop reference and device number */
    DvmType loop_ref = *pLoopRef;
    DvmType device_num = dvmh_loop_get_device_num_C(loop_ref);
    /* Parameters */
    dvmh_loop_autotransform_C(loop_ref, A_hdr);
    double *A = (double *)dvmh_get_natural_base_C(device_num, A_hdr);
    DvmType A_devHdr[64];
    tmpVar = dvmh_fill_header_C(device_num, A, A_hdr, A_devHdr, 0);
    assert(tmpVar == 0 || tmpVar == 1);
    dvmh_loop_autotransform_C(loop_ref, B_hdr);
    double *B = (double *)dvmh_get_natural_base_C(device_num, B_hdr);
    DvmType B_devHdr[64];
    tmpVar = dvmh_fill_header_C(device_num, B, B_hdr, B_devHdr, 0);
    assert(tmpVar == 0 || tmpVar == 1);
    /* Supplementary variables for loop handling */
    DvmType boundsLow[1], boundsHigh[1], loopSteps[1];
    DvmType blocksS[1];
    DvmType restBlocks;
    dim3 blocks(1, 1, 1), threads(0, 0, 0);
    hipStream_t stream;

    /* Choose index type for CUDA kernel */
    int kernelIndexT = dvmh_loop_guess_index_type_C(loop_ref);
    if (kernelIndexT == rt_LONG) kernelIndexT = (sizeof(long) <= sizeof(int) ? rt_INT : rt_LLONG);
    assert(kernelIndexT == rt_INT || kernelIndexT == rt_LLONG);

    /* Get CUDA configuration parameters */
    extern DvmType loop_jac1d_50_cuda_kernel_int_regs;
    if (kernelIndexT == rt_INT) dvmh_loop_cuda_get_config_C(loop_ref, 0, loop_jac1d_50_cuda_kernel_int_regs, &threads, &stream, 0);
    extern DvmType loop_jac1d_50_cuda_kernel_llong_regs;
    if (kernelIndexT == rt_LLONG) dvmh_loop_cuda_get_config_C(loop_ref, 0, loop_jac1d_50_cuda_kernel_llong_regs, &threads, &stream, 0);

    /* Calculate computation distribution parameters */
    dvmh_loop_fill_bounds_C(loop_ref, boundsLow, boundsHigh, loopSteps);
    blocksS[0] = ((boundsHigh[0] - boundsLow[0] + loopSteps[0]) / loopSteps[0] + (threads.x - 1)) / threads.x;

    /* Reductions-related stuff */
    double eps;
    double *eps_grid;
    dvmh_loop_cuda_register_red_C(loop_ref, 1, (void **)&eps_grid, 0);
    dvmh_loop_red_init_C(loop_ref, 1, &eps, 0);
    dvmh_loop_cuda_red_prepare_C(loop_ref, 1, blocksS[0] * threads.x * threads.y * threads.z, 0);

    /* GPU execution */
    restBlocks = blocksS[0];
    while (restBlocks > 0) {
        blocks.x = (restBlocks <= 65535 ? restBlocks : (restBlocks <= 65535 * 2 ? restBlocks / 2 : 65535));
        if (kernelIndexT == rt_INT) loop_jac1d_50_cuda_kernel_int<<<blocks, threads, 0, stream>>>(A, B, boundsLow[0], boundsHigh[0], eps, eps_grid, blocksS[0] - restBlocks);
        if (kernelIndexT == rt_LLONG) loop_jac1d_50_cuda_kernel_llong<<<blocks, threads, 0, stream>>>(A, B, boundsLow[0], boundsHigh[0], eps, eps_grid, blocksS[0] - restBlocks);
        restBlocks -= blocks.x;
    }

    dvmh_loop_cuda_red_finish_C(loop_ref, 1);
}

__global__ void loop_jac1d_60_cuda_kernel_int(double * DVMH_RESTRICT A, double * DVMH_RESTRICT B, int boundsLow_1, int boundsHigh_1, int blockOffset) {
    /* Parameters */
    /* Supplementary variables for loop handling */
    int restBlocks, curBlocks;
    /* User variables - loop index variables and other private variables */
    int i;

    restBlocks = blockIdx.x + blockOffset;
    curBlocks = restBlocks;
    i = boundsLow_1 + (curBlocks * blockDim.x + threadIdx.x);
    if (i <= boundsHigh_1)
    {
        do
        {
            B[i] = (A[i - 1] + A[i + 1]) / 2.;
        }
        while(0);
    }
}
__global__ void loop_jac1d_60_cuda_kernel_llong(double * DVMH_RESTRICT A, double * DVMH_RESTRICT B, long long boundsLow_1, long long boundsHigh_1, long long blockOffset) {
    /* Parameters */
    /* Supplementary variables for loop handling */
    long long restBlocks, curBlocks;
    /* User variables - loop index variables and other private variables */
    int i;

    restBlocks = blockIdx.x + blockOffset;
    curBlocks = restBlocks;
    i = boundsLow_1 + (curBlocks * blockDim.x + threadIdx.x);
    if (i <= boundsHigh_1)
    {
        do
        {
            B[i] = (A[i - 1] + A[i + 1]) / 2.;
        }
        while(0);
    }
}

extern "C" void loop_jac1d_60_cuda(DvmType *pLoopRef, DvmType A_hdr[], DvmType B_hdr[]) {
    DvmType tmpVar;
    /* Loop reference and device number */
    DvmType loop_ref = *pLoopRef;
    DvmType device_num = dvmh_loop_get_device_num_C(loop_ref);
    /* Parameters */
    dvmh_loop_autotransform_C(loop_ref, A_hdr);
    double *A = (double *)dvmh_get_natural_base_C(device_num, A_hdr);
    DvmType A_devHdr[64];
    tmpVar = dvmh_fill_header_C(device_num, A, A_hdr, A_devHdr, 0);
    assert(tmpVar == 0 || tmpVar == 1);
    dvmh_loop_autotransform_C(loop_ref, B_hdr);
    double *B = (double *)dvmh_get_natural_base_C(device_num, B_hdr);
    DvmType B_devHdr[64];
    tmpVar = dvmh_fill_header_C(device_num, B, B_hdr, B_devHdr, 0);
    assert(tmpVar == 0 || tmpVar == 1);
    /* Supplementary variables for loop handling */
    DvmType boundsLow[1], boundsHigh[1], loopSteps[1];
    DvmType blocksS[1];
    DvmType restBlocks;
    dim3 blocks(1, 1, 1), threads(0, 0, 0);
    hipStream_t stream;

    /* Choose index type for CUDA kernel */
    int kernelIndexT = dvmh_loop_guess_index_type_C(loop_ref);
    if (kernelIndexT == rt_LONG) kernelIndexT = (sizeof(long) <= sizeof(int) ? rt_INT : rt_LLONG);
    assert(kernelIndexT == rt_INT || kernelIndexT == rt_LLONG);

    /* Get CUDA configuration parameters */
    extern DvmType loop_jac1d_60_cuda_kernel_int_regs;
    if (kernelIndexT == rt_INT) dvmh_loop_cuda_get_config_C(loop_ref, 0, loop_jac1d_60_cuda_kernel_int_regs, &threads, &stream, 0);
    extern DvmType loop_jac1d_60_cuda_kernel_llong_regs;
    if (kernelIndexT == rt_LLONG) dvmh_loop_cuda_get_config_C(loop_ref, 0, loop_jac1d_60_cuda_kernel_llong_regs, &threads, &stream, 0);

    /* Calculate computation distribution parameters */
    dvmh_loop_fill_bounds_C(loop_ref, boundsLow, boundsHigh, loopSteps);
    blocksS[0] = ((boundsHigh[0] - boundsLow[0] + loopSteps[0]) / loopSteps[0] + (threads.x - 1)) / threads.x;

    /* GPU execution */
    restBlocks = blocksS[0];
    while (restBlocks > 0) {
        blocks.x = (restBlocks <= 65535 ? restBlocks : (restBlocks <= 65535 * 2 ? restBlocks / 2 : 65535));
        if (kernelIndexT == rt_INT) loop_jac1d_60_cuda_kernel_int<<<blocks, threads, 0, stream>>>(A, B, boundsLow[0], boundsHigh[0], blocksS[0] - restBlocks);
        if (kernelIndexT == rt_LLONG) loop_jac1d_60_cuda_kernel_llong<<<blocks, threads, 0, stream>>>(A, B, boundsLow[0], boundsHigh[0], blocksS[0] - restBlocks);
        restBlocks -= blocks.x;
    }
}

extern "C" void initCdvmhGlobals_jac1d_410751730();
extern "C" void finishCdvmhGlobals_jac1d_1316048831();
static DvmhModuleInitializer initCdvmhGlobals_jac1d_410751730_initializer(initCdvmhGlobals_jac1d_410751730, finishCdvmhGlobals_jac1d_1316048831);

